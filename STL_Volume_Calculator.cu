#include "hip/hip_runtime.h"
//	author: Benedykt Bela

#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>



//	funkcja na GPU sumuj�ca ze sob� dwa wektory danych 
__global__ void add(float* R_side, float* L_side, int size,  float accuracy)
{
    int i = threadIdx.x;

	R_side[i] = R_side[i] - L_side[i];
	R_side[i] = R_side[i] * accuracy * accuracy;
}



//	funkcja na GPU s�u��ca do podzielenia odcinka na punkty zgodnie z zadan� dok�adno�ci�
//	dla kolejnych warto�ci zmiennej y punkt przeci�cia prostej y i odcinka mo�emy szuka� naraz
__global__ void crossline_cuda(float* y0, float* y1, float* x0, float* x1, char* strona, float* y)
{
	struct point2D		
	{
		float x;
		float y;
	};


	struct direct_point
	{
		point2D point;
		char strona;
	};


	//	tworz� nowy punkt przeci�cia odcinka z prost� y
	direct_point* punkt = new direct_point;
	point2D pomocniczy;		//	dodatkowy punkt usprawnia obliczenia


	int i = threadIdx.x;

	//	je�eli odcinek ko�czy si� i zaczyna w tym samym punkcie
	if (y0[i] == y1[i] && x0[i] == x1[i])
		punkt[0].strona = 'P';
	else
	{
		//	szukanie przeci�cia prostej y z zadanym odcinkiem
		pomocniczy.x = (x0[i] - x1[i]) * (y[i] - y1[i]) / (y1[i] - y0[i]);
		punkt[0].point.x = x1[i] + pomocniczy.x;
		punkt[0].strona = strona[i];
		punkt[0].point.y = y[i];
	}


	//	je�eli znaleziony punkt jest punktem skrajnym odcinka
	if (y0[i] < y1[i] && y1[i] == punkt[0].point.y)
		punkt[0].strona = 'P';
	if (y0[i] > y1[i] && y0[i] == punkt[0].point.y)
		punkt[0].strona = 'P';


	//	do wektora, kt�ry skopiuj� na CPU zapisuj� dane znalezionego punktu preci�cia
	y0[i] = punkt[0].point.y;
	x0[i] = punkt[0].point.x;
	strona[i] = punkt[0].strona;

}



//	funkcja na GPU s�u��ca do podzia�u tr�jk�ta na odcinki poprzec robienie przekroju 
//	przesuwaj�c� si� p�aszczyzn�
__global__ void crossection_cuda(float* y0, float* y1, float* y2, float* x0, 
								float* x1, float* x2, float* z0, 
								float* z1, float* z2, char* strona, float* z, float* normx)
{
	struct point3D
	{
		float x;
		float y;
		float z;
	};


	struct point2D
	{
		float x;
		float y;
	};


	struct line
	{
		point2D point[2];
		char strona;
	};


	//	tworz� struktury linia oraz punkt w przestrzeni tr�jwymiarowej, 
	//	kt�re b�d� pomocne przy dalszych obliczeniach
	line* linia = new line();
	point3D pomocniczyp;

	int j = 0;		//	potrzebne do zliczania kt�ry punkt aktualnie zapisuj� do struktury linia
	int i = threadIdx.x;

	
	//	poni�sze instrukcje warunkowe sprawdzaj� mi�dzy kt�rymi punktami znajduje si� obecnie 
	//	przeszukiwana p�aszczyzna, czyli kt�re odcinki tr�jk�ta b�dziemy przecina� oraz te odcinki
	//	przecina
	if ((z1[i] >= z[i] > z0[i] || z1[i] < z[i] <= z0[i]) && ((z0[i] - z1[i]) != 0))
	{
		pomocniczyp.x = x0[i] - x1[i];
		pomocniczyp.y = y0[i] - y1[i];
		pomocniczyp.x = pomocniczyp.x * (z[i] - z1[i]) / (z0[i] - z1[i]);
		pomocniczyp.y = pomocniczyp.y * (z[i] - z1[i]) / (z0[i] - z1[i]);
		linia[0].point[j].x = x1[i] + pomocniczyp.x;
		linia[0].point[j].y = y1[i] + pomocniczyp.y;

		j++;
	}

	if ((z1[i] >= z[i] > z2[i] || z1[i] < z[i] <= z2[i]) && (z2[i] - z1[i]) != 0)
	{
		pomocniczyp.x = x2[i] - x1[i];
		pomocniczyp.y = y2[i] - y1[i];
		pomocniczyp.x = pomocniczyp.x * (z[i] - z1[i]) / (z2[i] - z1[i]);
		pomocniczyp.y = pomocniczyp.y * (z[i] - z1[i]) / (z2[i] - z1[i]);
		linia[0].point[j].x = x1[i] + pomocniczyp.x;
		linia[0].point[j].y = y1[i] + pomocniczyp.y;

		j++;
	}

	if ((z0[i] >= z[i] > z2[i] || z0[i] < z[i] <= z2[i]) && (z2[i] - z0[i]) != 0)
	{
		pomocniczyp.x = x2[i] - x0[i];
		pomocniczyp.y = y2[i] - y0[i];
		pomocniczyp.x = pomocniczyp.x * (z[i] - z0[i]) / (z2[i] - z0[i]);
		pomocniczyp.y = pomocniczyp.y * (z[i] - z0[i]) / (z2[i] - z0[i]);
		linia[0].point[j].x = x0[i] + pomocniczyp.x;
		linia[0].point[j].y = y0[i] + pomocniczyp.y;

		j++;
	}


	//	sprawdzam w kt�r� stron� jest skierowana normalna danego tr�jk�ta, �eby wiedzie�
	//	gdzie jest �rodek badanego obiektu, a gdzie strona zewn�trzna
	if (normx[i] > 0)
		linia[0].strona = 'R';		//	R - z prawej
	else if (normx[i] < 0)
		linia[0].strona = 'L';		//	L - z lewej
	else
		linia[0].strona = 'T';		//	T oznacza tr�jk�t prostopad�y do osi y


	//	zapisuj� znalezione dane do wektor�w, kt�re skopiuj� na CPU
	strona[i] = linia[0].strona;
	x0[i] = linia[0].point[0].x;
	x1[i] = linia[0].point[1].x;
	y0[i] = linia[0].point[0].y;
	y1[i] = linia[0].point[1].y;

}



using namespace std;


struct point3D;
struct point2D;
struct triangles;
struct line;
struct plane2D;
class Wczytywanie;
class Szukaj;


struct point3D
{
	float x;
	float y;
	float z;
};


struct point2D
{
	float x;
	float y;
};


//	ka�dy tr�jk�t sk�ada si� z trzech punkt�w oraz wektora normalnego okre�laj�cego
//	kierunek wn�trza figury
struct triangles
{
	point3D normalna;
	point3D first;
	point3D second;
	point3D third;
};


struct line
{
	point2D point[2];
	char strona;		//	strona oznacza kierunek wn�trza figury, kt�rej fragmentem
						//	jest dany odcinek
};


struct direct_point
{
	point2D point;
	char strona;
};


//	plane2D zawiera jedynie wektor sk�adaj�cy si� ze struktur linia
//	ta struktura umo�liwia zrobienie swego rodzaju tablicy wektor�w
struct plane2D
{
	vector<line> plane;
};


//	tablica wektor�w zawieraj�ca ostateczne punkty do obliczania obj�to�ci bry�y
struct final_points
{
	vector<direct_point> punkt;
};



//	klasa s�u��ca do wczytywania danych z pliku oraz przechowywania takich informacji
//	jak globalne minimum oraz maksimum w ka�dej p�aszczy�nie, ilo�� pobranych 
//	tr�jk�t�w, dok�adno�� oblcicze� oraz funkcj� do wypisywania posiadanych danych
class Wczytywanie {
public:
	Wczytywanie();

	int ile = 0;
	float accuracy = 0.1;
	float min_z = 0;
	float min_x = 0;
	float min_y = 0;
	float max_y = 0;
	float max_z = 0;
	float max_x = 0;
	triangles *triangle;
	void wypisz_dane();


private:

};



//	g��wna klasa programu zawieraj�ca odpowiednie funkcje obliczaj�ce obj�to�� figury
class Szukaj {
public:
	Szukaj();

	//	wewn�trz g��wnej klasy programu inicjuj� instancj� klasy Wczytywanie, aby mie� 
	//	dost�p do jej danych 
	Wczytywanie wczytywanie;
	void cut_all_triangles();
	void cut_all_lines();
	void cut_triangle(triangles triangle);
	void cut_line(line linia, float g_min, float g_max);
	void wypisz_plane();
	float objetosc = 0;		//	zmienna zawieraj�ca warto�� obj�to�ci naszego obiektu


private:
	int ile_warstw;		//	liczba warstw przeci�ciu obiektu przesuwaj�c� si� p�aszczyzn� z
	float global_min = wczytywanie.min_z;		//	globalne minimum obiektu w osi z
	//	generalnie zmienne wykorzstywane w innych miejscach programu
	plane2D* plane;		
	point3D* results;
	float accuracy = wczytywanie.accuracy;
	int plane_size;
	final_points* final_point;

};



//	g��wna funkcja programu
int main()
{
	
	Szukaj szukaj;		//	inicjalizuj� instancj� klasy Szukaj
	szukaj.cut_all_triangles();		//	wykonuj� poci�cie wszystkich tr�jk�t�w
	szukaj.wypisz_plane();		//	wypisuje odcinki, kt�re sk�adaj� si� na przekroje obiektu
	szukaj.cut_all_lines();		//	przecina wszystkie dostepne linie i znajduje punkty przeci�cia

	cout << endl << szukaj.objetosc << endl << endl;		//	wypisuj� znalezion� obj�to�� figury

}



//	poni�sza funkcja przecina wszystkie linie tworz�ce przekroje r�wnoleg�e do p�aszczyzny z
//	i znajduje punkty przeci�cia na podstawie kt�rych oblicza obj�to�� bry�y
void Szukaj::cut_all_lines()
{
	float min, max;


	//	wykonuj� ci�cie i liczenie dla ka�dego przekroju r�wnoleg�ego do p�aszczyzny z
	for (int i = 0; i < plane_size; i++)
	{
		//	ustawiam pocz�tkow� warto�� dla zmiennych min i max
		min = plane[i].plane[0].point[0].y;
		max = min;

		//	dla ka�dego przekroju sprawdzam jego wsp�rz�dn� maksymaln� i minimaln� w osi y
		for (int j = 0; j < plane[i].plane.size(); j++)
		{
			if (min > plane[i].plane[j].point[0].y)
				min = plane[i].plane[j].point[0].y;
			if (min > plane[i].plane[j].point[1].y)
				min = plane[i].plane[j].point[1].y;
			if (max < plane[i].plane[j].point[0].y)
				max = plane[i].plane[j].point[0].y;
			if (max < plane[i].plane[j].point[1].y)
				max = plane[i].plane[j].point[1].y;
		}

		//	znalezione warto�ci min i max zaokr�glam do najbli�szej warto�ci y, kt�ra
		//	jest wielokrotno�ci� dok�adno�ci
		min = min / accuracy;
		min = ceil(min);
		min = min * accuracy;

		max = max / accuracy;
		max = floor(max);
		max = max * accuracy;

		//	tworz� now� tablic� punkt�w ostatecznych 
		final_point = new final_points[int((max - min) / accuracy + 1)];

		//	ka�d� lini� w danym przekroju przecinam i generuj� ostateczne punkty do obliczenia obj�to�ci
		for (int j = 0; j < plane[i].plane.size(); j++)
			cut_line(plane[i].plane[j], min, max);			

		int pomocnicza = int((max - min) / accuracy + 1);		//	liczba danych w tej iteracji p�tli
		
		//	wektory na CPU i GPU z warto�ciami do oblicze�
		float* R_side = new float[pomocnicza];
		float* d_R_side = new float[pomocnicza];
		float* L_side = new float[pomocnicza];
		float* d_L_side = new float[pomocnicza];

		//	wszystkie dane przegl�damy i zapisujemy te ze stron� 'R' do jedgeno wektora, 
		//	a te ze stron� 'L' do drugeigo wektora
		for (int k = 0; k < int((max - min) / accuracy); k++)
		{
			for (int l = 0; l < final_point[k].punkt.size(); l++)
			{
				if (final_point[k].punkt[l].strona == 'R')
					R_side[k] = final_point[k].punkt[l].point.x;

				else if (final_point[k].punkt[l].strona == 'L')
					L_side[k] = final_point[k].punkt[l].point.x;

			}

			cout << R_side[k] << " " << L_side[k] << " ";

			cout << endl;
		}

		//	alokuj� miejsce na GPU
		hipMalloc(& d_R_side, pomocnicza * sizeof(float));
		hipMalloc(& d_L_side, pomocnicza * sizeof(float));

		//	oraz kopiuj� potrzebne dane
		hipMemcpy(d_R_side, R_side, pomocnicza * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_L_side, L_side, pomocnicza * sizeof(float), hipMemcpyHostToDevice);

		//	wykonuj� funkcj� dodawanie na GPU
		add << <1, pomocnicza >> > (d_R_side, d_L_side, pomocnicza, accuracy);

		//	i kopiuj� na CPU potrzebne dane
		hipMemcpy(R_side, d_R_side, pomocnicza * sizeof(float), hipMemcpyDeviceToHost);

		//	sumuj� skopiowane dane i dodaje do zmiennej obj�to��
		for (int j = 0; j < pomocnicza; j++)
			objetosc += R_side[j];

		//	usuwam dynamicznie alokowan� tablic� zmiennych 
		delete[] final_point;

	}
}



//	funkcja, kt�ra pobiera dane jednego obiektu line oraz globalne minimum i globalne maksimum,
//	a  zwraca punkty, kt�re s� punktami przeci�cia tej linii z p�aszczyznami prostopad�ymi do osi OY
//	ustawionymi w odleg�o�ci od siebie wynosz�cej warto�� zmiennej accuracy
void Szukaj::cut_line(line linia, float g_min, float g_max)
{
	float min, max;

	//	ustalam kt�ry koniec linii ma wi�ksz� wsp�rz�dn� y i przypisuj� odpowiednio do zmiennych min i max
	if (linia.point[0].y > linia.point[1].y)
	{
		min = linia.point[1].y;
		max = linia.point[0].y;
	}
	else
	{
		min = linia.point[0].y;
		max = linia.point[1].y;
	}

	//	zaokr�glam warto�ci min i max w tej spos�b, �e nowe min jest najbli�sz�, ale wi�ksz� wielokrotno�ci� 
	//	zmiennej accuracy, natomiast max jest najbli�sz�, ale mniejsz� wielokrotno�ci� zmiennej accuracy
	min = min / accuracy;
	min = ceil(min);
	min = min * accuracy;

	max = max / accuracy;
	max = floor(max);
	max = max * accuracy;

	//	zmienne pomocnicze s�u��ce do oblicze�
	float yorg = min;
	float y_help = yorg;
	int index = 0;
	int ile = 0;

	//	zliczam ile wielokrotno�ci zmiennej accuracy mie�ci si� mi�dzy warto�ciami min i max,
	//	poniewa� tyle punkt�w funkcja musi wygenerowa�
	while (y_help <= max)
	{
		y_help += accuracy;
		ile++;
	}

	//	wektory zmiennych, do kt�rych zapisuj� odpowiednie warto�ci, kt�re p�niej skopiuj� na GPU i wykorzystam
	float* y0 = new float[ile];
	float* y1 = new float[ile];
	float* x0 = new float[ile];
	float* x1 = new float[ile];
	float* y = new float[ile];
	char* strona = new char[ile];

	//	odpowiedniki powy�szych wska�nik�w na GPU
	float* dy0 = new float[ile];
	float* dy1 = new float[ile];
	float* dx0 = new float[ile];
	float* dx1 = new float[ile];
	float* dy = new float[ile];
	char* dstrona = new char[ile];

	
	//	nadaj� zainicjowanym powy�ej wektorom odpowiednie warto�ci ze struktury linia
	for (int i = 0; i < ile; i++)
	{
		y0[i] = linia.point[0].y;
		y1[i] = linia.point[1].y;
		x0[i] = linia.point[0].x;
		x1[i] = linia.point[1].x;
		strona[i] = linia.strona;
		y[i] = yorg; 
		yorg += accuracy;
	}

	//	alokuj� pami�� na GPU
	hipMalloc(&dy0, ile * sizeof(float));
	hipMalloc(&dy1, ile * sizeof(float));
	hipMalloc(&dx0, ile * sizeof(float));
	hipMalloc(&dx1, ile * sizeof(float));
	hipMalloc(&dy, ile * sizeof(float));
	hipMalloc(&dstrona, ile * sizeof(char));

	//	i kopiuj� na GPU dane z CPU
	hipMemcpy(dy0, y0, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dy1, y1, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dx0, x0, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dx1, x1, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dy, y, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dstrona, strona, ile * sizeof(char), hipMemcpyHostToDevice);

	//	wywo�anie funkcji wykonywanej na GPU, kt�ra zosta�a opisana w miejscu definicji
	crossline_cuda << <1, ile >> > (dy0, dy1, dx0, dx1, dstrona, dy);

	//	kopiuj� dane z GPU z powrotem na CPU
	hipMemcpy(y0, dy0, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y1, dy1, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(x0, dx0, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(x1, dx1, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y, dy, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(strona, dstrona, ile * sizeof(char), hipMemcpyDeviceToHost);

	//	zwalniam pami�� na GPU
	hipFree(dy0);
	hipFree(dy1);
	hipFree(dx0);
	hipFree(dx1);
	hipFree(dy);
	hipFree(dstrona);

	//	ustawiam ponownie warto�� yorg na warto�� wcze�niej obliczonego min
	yorg = min;


	//	dla ka�dego wyci�tego punktu dodaj� go do tablicy punkt�w ostatecznych 
	for (int i = 0; i < ile; i++)
	{
		//	poni�sze kilka instrukcji ustala indeks w tablicy punkt�w ostatecznych do kt�rego dany punkt ma
		//	zosta� przypisany
		float help = yorg - g_min;

		if (help < 0)
			help = 1;

		index = int(help / accuracy);

		//	tworz� nowy wska�nik na obiekt direct_point
		direct_point* abc = new direct_point;

		//	przypisuj� do obiektu abc odpowiednie warto�ci z wektor�w danych obliczonych na GPU
		abc[0].point.x = x0[i];
		abc[0].point.y = y0[i];
		abc[0].strona = strona[i];

		//	je�eli strona danego punktu nie ma warto�ci 'P', to dodajemy ten punkt do wektora pod
		//	odpowiednim adresem tablicy final_point
		if (abc[0].strona != 'P')
			final_point[index].punkt.push_back(abc[0]);

		//	zwi�kszam aktualn� warto�� y, �eby nast�pny punkt zapisa� pod innym indeksem w 
		//	tablicy final_point, bo zmienna index jest ustalana w�a�nie na podstawie yorg
		yorg += accuracy;

		//	usuwam dynamicznie alokowan� zmienn� abc
		delete abc;
	}
}



//	funkcja nale�y do klasy Szukaj i s�u�y do wypisania w konsoli linii tworz�cych przekroje na poszczeg�lnych
//	p�aszczyznach prostopad�ych do osi OZ
void Szukaj::wypisz_plane()
{
	for (int i = 0; i < ile_warstw; i++)
	{
		int k = plane[i].plane.size();

		for (int j = 0; j < k; j++)
		{
			cout << plane[i].plane[j].point[0].x << " ";
			cout << plane[i].plane[j].point[0].y << "     ";
			cout << plane[i].plane[j].point[1].x << " ";
			cout << plane[i].plane[j].point[1].y << "     ";
			cout << endl;
		}

		cout << endl;
	}
}



//	ta kr�tka funkcja po prostu wywo�uje funkcj� cut_triangle dla wszystkich tr�jk�t�w wczytanych do programu
void Szukaj::cut_all_triangles()
{
	for (int i = 0; i < wczytywanie.ile; i++)
		cut_triangle(wczytywanie.triangle[i]);

}



//	poni�sza funkcja zwraca linie, kt�re powstaj� na skutek przeci�cia zadanego tr�jk�ta przez p�aszczyzn�
//	OXY przesuwan� z krokiem accuracy wzd�u� osi Z
void Szukaj::cut_triangle(triangles triangle)
{
	float min, max;

	//	ustalam minimaln� i maksymaln� wsp�rz�dn� tr�jk�ta w osi Z
	if (triangle.first.z >= triangle.second.z && triangle.first.z >= triangle.third.z)
		max = triangle.first.z;
	else if (triangle.second.z >= triangle.second.z && triangle.second.z >= triangle.third.z)
		max = triangle.second.z;
	else if (triangle.third.z >= triangle.second.z && triangle.third.z >= triangle.first.z)
		max = triangle.third.z;

	if (triangle.first.z <= triangle.second.z && triangle.first.z <= triangle.third.z)
		min = triangle.first.z;
	else if (triangle.second.z <= triangle.second.z && triangle.second.z <= triangle.third.z)
		min = triangle.second.z;
	else if (triangle.third.z <= triangle.second.z && triangle.third.z <= triangle.first.z)
		min = triangle.third.z;

	//	zaokr�glam zmienn� min do najbli�szej, wi�kszej od zmiennej min wielokrotno�ci zmiennej accuracy
	min = min / accuracy;
	min = ceil(min);
	min = min * accuracy;

	//	zaokr�glam zmienn� max do najbli�szej, mniejszej od zmiennej max wielokrotno�ci zmiennej accuracy
	max = max / accuracy;
	max = floor(max);
	max = max * accuracy;

	//	zmienne pomocnicze 
	int ile = 0;
	float zorg = min;
	float z_help = zorg;

	//	sprawdzam ile wielokrotno�ci zmiennej accuracy zmie�ci si� mi�dzy min i max, bo tyle linii 
	//	funkcja wytnie z tr�jk�ta
	while (z_help <= max)
	{
		z_help += accuracy;
		ile++;
	}

	//	wektory zmiennych do kt�rych zapisuj� odpowiednie dane, kt�re zostan� skopiowane na GPU
	//	i na ich podstawie GPU dokona oblicze�
	float* y0 = new float[ile];
	float* y1 = new float[ile];
	float* y2 = new float[ile];
	float* x0 = new float[ile];
	float* x1 = new float[ile];
	float* x2 = new float[ile];
	float* z0 = new float[ile];
	float* z1 = new float[ile];
	float* z2 = new float[ile];
	float* z = new float[ile];
	float* normx = new float[ile];
	char* strona = new char[ile];

	//	odpowiedniki powy�szych wska�nik�w wykorzystywane na GPU
	float* dy0 = new float[ile];
	float* dy1 = new float[ile];
	float* dy2 = new float[ile];
	float* dx0 = new float[ile];
	float* dx1 = new float[ile];
	float* dx2 = new float[ile];
	float* dz0 = new float[ile];
	float* dz1 = new float[ile];
	float* dz2 = new float[ile];
	float* dz = new float[ile];
	float* dnormx = new float[ile];
	char* dstrona = new char[ile];


	//	przypisuj� zadeklarowanym wektorom odpowiednie dane 
	for (int i = 0; i < ile; i++)
	{
		y0[i] = triangle.first.y;
		y1[i] = triangle.second.y;
		y2[i] = triangle.third.y;
		x0[i] = triangle.first.x;
		x1[i] = triangle.second.x;
		x2[i] = triangle.third.x;
		z0[i] = triangle.first.z;
		z1[i] = triangle.second.z;
		z2[i] = triangle.third.z;
		normx[i] = triangle.normalna.x;
		z[i] = zorg;
		zorg += accuracy;
	}

	//	alokuj� pami�� na GPU
	hipMalloc(&dy0, ile * sizeof(float));
	hipMalloc(&dy1, ile * sizeof(float));
	hipMalloc(&dy2, ile * sizeof(float));
	hipMalloc(&dz0, ile * sizeof(float));
	hipMalloc(&dz1, ile * sizeof(float));
	hipMalloc(&dz2, ile * sizeof(float));
	hipMalloc(&dx0, ile * sizeof(float));
	hipMalloc(&dx1, ile * sizeof(float));
	hipMalloc(&dx2, ile * sizeof(float));
	hipMalloc(&dz, ile * sizeof(float));
	hipMalloc(&dnormx, ile * sizeof(float));
	hipMalloc(&dstrona, ile * sizeof(char));
	
	//	i kopiuj� potrzebne dane na GPU
	hipMemcpy(dy0, y0, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dy1, y1, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dy2, y2, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dx0, x0, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dx1, x1, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dx2, x2, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dz0, z0, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dz1, z1, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dz2, z2, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dz, z, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dnormx, normx, ile * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dstrona, strona, ile * sizeof(char), hipMemcpyHostToDevice);
	
	//	funkcja na GPU opisana w miejscu definicji
	crossection_cuda << <1, ile >> > (dy0, dy1, dy2, dx0, dx1, dx2, dz0, dz1, dz2, dstrona, dz, dnormx);

	//	i kopiuj� z powrotem na CPU wektory danych po wykonaniu funkcji na GPU
	hipMemcpy(y0, dy0, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y1, dy1, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y2, dy2, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(x0, dx0, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(x1, dx1, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(x2, dx2, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z0, dz0, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z1, dz1, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z2, dz2, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(z, dz, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(normx, dnormx, ile * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(strona, dstrona, ile * sizeof(char), hipMemcpyDeviceToHost);

	//	zwalniam pami�� na GPU
	hipFree(dy0);
	hipFree(dy1);
	hipFree(dy2);
	hipFree(dx0);
	hipFree(dx1);
	hipFree(dx2);
	hipFree(dz0);
	hipFree(dz1);
	hipFree(dz2);
	hipFree(dz);
	hipFree(dstrona);
	hipFree(dnormx);

	//	ustawiam warto�� zmiennej z z powrotem na warto�� znalezionego minimum
	zorg = min;
	int index = 0;


	//	dla ka�dej wyci�tej linii zapisuj� j� pod odpowiednim indeksem w tablicy plane
	for (int i = 0; i < ile; i++)
	{
		//	obliczam index pod kt�rym dana linia powinna zosta� zapisana
		index = int((zorg - min) / accuracy);

		if (index < 0)
			index = 0;

		//	tworz� wska�nik na now� lini�
		line* abc = new line;

		//	i przypisuj� do tej linii warto�ci obliczone na GPU
		abc[0].point[0].x = x0[i];
		abc[0].point[1].x = x1[i];
		abc[0].point[0].y = y0[i];
		abc[0].point[1].y = y1[i];
		abc[0].strona = strona[i];

		//	je�eli strona ma warto�� inn� ni� 'T', to dodaj� t� lini� do tablicy plane
		if (abc[0].strona != 'T')
			plane[index].plane.push_back(abc[0]);

		//	zwi�kszam zorg, aby nast�pna linia zosta�a zapisana pod innym indeksem,
		//	bo na podstawie zorg jest obliczany index
		zorg += accuracy;

		//	usuwam dynamicznie alokowan� zmienn� abc
		delete abc;

	}
}



//	konstruktor klasy Szukaj
Szukaj::Szukaj()
{
	//	obliczam ile warst b�dzie zawiera� program po przeci�ciu bry�y odpowiednimi p�aszczyznami 
	//	oddalonymi od siebie o warto�� zmiennej accuracy
	ile_warstw = (wczytywanie.max_z - wczytywanie.min_z) / wczytywanie.accuracy;
	plane_size = ile_warstw + 1;

	//	deklaruj� dwie tablice struktur do przechowywania danych na podstawie obliczonego powy�ej rozmiaru
	plane = new plane2D[plane_size];
	results = new point3D[plane_size];

}



//	konstruktor klasy Wczytywanie, kt�ry jest jednocze�nie g��wnym dzia�aniem wykonywanym przez t� klas�
Wczytywanie::Wczytywanie()
{
	//	pobieramy od u�ytkownika nazw� pliku oraz dok�adno�� wykonywanych oblicze�
	string word, nazwa_pliku;

	cout << "Podaj nazwe pliku wraz z rozszerzeniem:     ";
	cin >> nazwa_pliku;

	cout << endl << "Podaj dokladnosc w milimetrach:     ";
	cin >> accuracy;
	cout << endl << endl;

	//	tworz� instancj� klasy fstream i otwieram plik o zadanej wy�ej nazwie
	fstream solid;
	solid.open(nazwa_pliku, ios::in);

	float liczba;

	//	najpierw zliczam liczb� tr�jk�t�w zawartych w pliku, 
	//	�eby wiedzie� jak du�� stworzy� tablic� na dane
	while (solid.good() == true)
	{
		solid >> word;
		if (word == "normal")
			ile++;
	}

	//	tworz� tablic� strukt�r zawieraj�c� pobrane z pliku dane
	triangle = new triangles[ile];

	//	zamykam i otwieram ponownie przeszukiwany plik
	solid.close();
	solid.open(nazwa_pliku, ios::in);


	//	poni�sze instrukcje wynikaj� z budowy pliku formatu .stl, gdzie niekt�re s�owa pomijam i 
	//	wyci�gam tylko potrzebne do dzia�ania programu dane
	if (solid.good() == true)
	{
		solid >> word;
		solid >> word;
	}


	for (int i = 0; i < ile; i++)
	{
		for (int k = 0; k < 2; k++)
			solid >> word;

		solid >> liczba;
		triangle[i].normalna.x = liczba;
		solid >> liczba;
		triangle[i].normalna.y = liczba;
		solid >> liczba;
		triangle[i].normalna.z = liczba;

		for (int k = 0; k < 3; k++)
			solid >> word;

		solid >> liczba;
		triangle[i].first.x = liczba;
		solid >> liczba;
		triangle[i].first.y = liczba;
		solid >> liczba;
		triangle[i].first.z = liczba;

		solid >> word;
		solid >> liczba;
		triangle[i].second.x = liczba;
		solid >> liczba;
		triangle[i].second.y = liczba;
		solid >> liczba;
		triangle[i].second.z = liczba;

		solid >> word;
		solid >> liczba;
		triangle[i].third.x = liczba;
		solid >> liczba;
		triangle[i].third.y = liczba;
		solid >> liczba;
		triangle[i].third.z = liczba;

		for (int k = 0; k < 2; k++)
			solid >> word;

		if (i == 0)
		{
			min_z = max_z = triangle[i].first.z;
			min_y = max_y = triangle[i].first.y;
			min_x = max_x = triangle[i].first.x;
		}


		//	wyznaczanie najmniejszych i najwi�kszych warto�ci na poszczeg�lnych osiach
		if (triangle[i].first.z < min_z)
			min_z = triangle[i].first.z;
		else if (triangle[i].second.z < min_z)
			min_z = triangle[i].second.z;
		else if (triangle[i].third.z < min_z)
			min_z = triangle[i].third.z;
		else if (triangle[i].first.z > max_z)
			max_z = triangle[i].first.z;
		else if (triangle[i].second.z > max_z)
			max_z = triangle[i].second.z;
		else if (triangle[i].third.z > max_z)
			max_z = triangle[i].third.z;

		if (triangle[i].first.y < min_y)
			min_y = triangle[i].first.y;
		else if (triangle[i].second.y < min_y)
			min_y = triangle[i].second.y;
		else if (triangle[i].third.y < min_y)
			min_y = triangle[i].third.y;
		else if (triangle[i].first.y > max_y)
			max_y = triangle[i].first.y;
		else if (triangle[i].second.y > max_y)
			max_y = triangle[i].second.y;
		else if (triangle[i].third.y > max_y)
			max_y = triangle[i].third.y;

		if (triangle[i].first.x < min_x)
			min_x = triangle[i].first.x;
		else if (triangle[i].second.x < min_x)
			min_x = triangle[i].second.x;
		else if (triangle[i].third.x < min_x)
			min_x = triangle[i].third.x;
		else if (triangle[i].first.x > max_x)
			max_x = triangle[i].first.x;
		else if (triangle[i].second.x > max_x)
			max_x = triangle[i].second.x;
		else if (triangle[i].third.x > max_x)
			max_x = triangle[i].third.x;

	}

	
	//	zaokr�glam warto�� max_z do najbli�szej mniejszej wielokrotno�ci zmiennej accuracy
	max_z = max_z / accuracy;
	max_z = floor(max_z);
	max_z = max_z * accuracy;

	//	zaokr�glam warto�� min_z do najbli�szej wi�kszej wielokrotno�ci zmiennej accuracy
	min_z = min_z / accuracy;
	min_z = ceil(min_z);
	min_z = min_z * accuracy;

	//	zamykam plik z danymi
	solid.close();

}



//	funkcja s�u��ca do wypisania danych pobranych z pliku tekstowego formatu stl
void Wczytywanie::wypisz_dane()
{
	for (int i = 0; i < ile; i++)
	{
		cout << triangle[i].normalna.x << " " << triangle[i].normalna.y << " " << triangle[i].normalna.z << " "
			<< triangle[i].first.x << " " << triangle[i].first.y << " " << triangle[i].first.z << " "
			<< triangle[i].second.x << " " << triangle[i].second.y << " " << triangle[i].second.z << " "
			<< triangle[i].third.x << " " << triangle[i].third.y << " " << triangle[i].third.z << endl;
	}
}



